#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/Dispatch.h>
#include <ATen/Functions.h>
#include <ATen/ScalarOps.h>
#include <ATen/Tensor.h>
#include <ATen/autocast_mode.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/CUDAUtils.h>
#include <c10/cuda/CUDAGuard.h>
#include <torch/library.h>

#include "ComputeSparseTile.cuh"
#include "SparseSemiStructuredPack.cuh"
#include <hip/hip_runtime.h>

namespace torchao {

struct MetadataCutlass {
  // Layout needed to run 2:4 gemms in CUTLASS
  // There is basically a hardware specific value for every
  // 32x32 dense tile (1024 bits). Then these tiles are
  // stored in a Column-Major fashion
  ElementInputE *_meta;
  ElementInputE *_meta_trans;
  int64_t _meta_reordered_sy;
  int64_t _meta_trans_reordered_sx;

  // Define create_compressed_representation
  static std::tuple<at::Tensor, // return value of the function
                    at::Tensor, // packed
                    at::Tensor  // packed_meta
                    >
  create_compressed_representation(int rows, int cols, at::Tensor const &like) {
    TORCH_CHECK(like.scalar_type() == at::ScalarType::Half ||
                like.scalar_type() == at::ScalarType::BFloat16 ||
                like.scalar_type() == at::ScalarType::Float8_e4m3fn);
    auto roundedx = cutlass::round_up(rows, kWarpX);
    auto roundedy = cutlass::round_up(cols, kWarpY);

    // NB: Writing to `packed` tensors in transposed manner
    at::Tensor packed =
        at::empty({roundedx, cutlass::ceil_div(roundedy, 2)}, like.options());

    // hard code this for now to 16
    at::Tensor packed_meta =
        at::zeros({roundedx, 16}, like.options().dtype(at::ScalarType::Byte));
    return std::make_tuple(packed, packed, packed_meta);
  }

  // define get_meta_offset
  MetadataCutlass(at::Tensor metaN, at::Tensor metaT, int rows, int cols) {
    _meta = (ElementInputE *)metaN.data_ptr();
    _meta_reordered_sy = metaN.stride(0);
    _meta_trans = (ElementInputE *)metaT.data_ptr();
    _meta_trans_reordered_sx = metaT.stride(0);
  }
  CUTLASS_HOST_DEVICE
  int64_t _get_meta_offset(int warp_row, int thread_row, int warp_col,
                           int thread_col, int64_t total_rows) const {
    int64_t offset = 0;

    // warp handles a 4x128 chunk, so find the appropriate one
    offset += (warp_row / 4) * (total_rows) * 2;

    // Base offset for the warp's starting position
    // offset += warp_row * 4; // Each warp handles 4 rows
    // offset += (warp_col / 128) * stride; // Column offset for warps

    // // Thread position within the warp
    offset += thread_row * (total_rows / 2); // Each thread handles 1 row
    // offset += 128 * (thread_row / 4);

    // // Column offset within the warp
    // // Each thread handles 16 columns, and we're packing 2:4 sparsity
    offset += ((thread_col % 128) / 16) * 2;
    // offset += (thread_col / 32) * 4;

    return offset;
  }

  // Define get_metaN and get_metaT
  CUTLASS_HOST_DEVICE
  ElementInputE *get_metaN(int warp_row, int thread_row, int warp_col,
                           int thread_col) const {
    return _meta + _get_meta_offset(warp_row, thread_row, warp_col, thread_col,
                                    _meta_reordered_sy);
  }
};

template <typename KT, typename Metadata, typename Algorithm>
__global__ void __launch_bounds__(32 /* num_threads */, 20)
    sparse_semi_structured_tile_kernel(typename KT::Params p, Metadata metadata,
                                       Algorithm algo) {
  KT::sparse_semi_structured_tile_kernel(p, metadata, algo);
}

template <typename Element, typename MetadataFormat>
std::tuple<at::Tensor, at::Tensor, at::Tensor, at::Tensor, at::Tensor>
sparse_semi_structured_tile_typed(const at::Tensor input,
                                  std::string algorithm) {

  printf("sparse_semi_structured_tile_typed... \n");
  using KT = KernelTypes<Element>;
  std::optional<at::cuda::CUDAGuard> device_guard;
  if (!input.is_meta()) {
    device_guard.emplace(input.device());
  }

  TORCH_CHECK(input.dim() == 2, "Can only sparsify 2d tensors");
  TORCH_CHECK(input.stride(1) == 1, "Can only sparsify contiguous tensors. "
                                    "Sparsify the transpose otherwise.");

  auto rows = input.size(0);
  auto cols = input.size(1);

  auto [compressed, packed, packed_meta_reordered] =
      MetadataFormat::create_compressed_representation(rows, cols, input);
  auto [compressed_trans, packed_trans, packed_trans_meta_reordered] =
      MetadataFormat::create_compressed_representation(cols, rows, input);
  TORCH_CHECK(input.size(1) % 32 == 0,
              "Number of cols should be multiple of 32");

  typename KT::Params p;
  p.input = (Element const *)input.data_ptr();
  p.input_s0 = input.stride(0);
  p.input_dim0 = input.size(0);
  p.input_dim1 = input.size(1);

  p.packed = (Element *)packed.data_ptr();
  p.packed_stride = packed.stride(0);
  p.packed_trans = (Element *)packed_trans.data_ptr();
  p.packed_trans_stride = packed_trans.stride(0);

  MetadataFormat metadata = MetadataFormat(
      packed_meta_reordered, packed_trans_meta_reordered, rows, cols);
  at::Tensor threads_masks = at::empty(
      {p.getBlocksGrid().x * p.getThreadsGrid().x,
       p.getBlocksGrid().y * p.getThreadsGrid().y, sizeof(p.threads_masks[0])},
      input.options().dtype(at::ScalarType::Byte));
  p.threads_masks = (uint64_t *)threads_masks.data_ptr();

  printf("launching kernel ... \n");
  bool kernel_launched = false;
  auto launchKernel = [&](auto algo, std::string const &algo_name) {
    if (algo_name == algorithm) {
      kernel_launched = true;
      if (input.is_meta()) {
        return;
      }
      size_t smem_bytes = 0;
      sparse_semi_structured_tile_kernel<KT>
          <<<p.getBlocksGrid(), p.getThreadsGrid(), smem_bytes,
             at::cuda::getCurrentCUDAStream()>>>(p, metadata, algo);
    }
  };
  named_algorithms(launchKernel);
  TORCH_CHECK(kernel_launched, "Unknown algorithm \"", algorithm, "\"");
  C10_CUDA_KERNEL_LAUNCH_CHECK();
  return std::make_tuple(compressed, packed_meta_reordered, compressed_trans,
                         packed_trans_meta_reordered, threads_masks);
}

// <packed, packed_meta_reordered, packed_trans, packed_trans_meta_reorderd,
// threads_masks>
std::tuple<at::Tensor, at::Tensor, at::Tensor, at::Tensor, at::Tensor>
_sparse_semi_structured_tile(const at::Tensor &input,
                             std::string_view algorithm, bool use_cutlass) {
  std::string algo(algorithm.data(), algorithm.size());

  printf("Start debugging here ...\n");

  auto runTyped = [&](auto type) {
    using ElementT = decltype(type);
    return sparse_semi_structured_tile_typed<ElementT, MetadataCutlass>(input,
                                                                        algo);
  };

  if (input.scalar_type() == at::ScalarType::Half) {
    return runTyped(cutlass::half_t());
  } else if (input.scalar_type() == at::ScalarType::Float8_e4m3fn) {
    return runTyped(cutlass::float_e4m3_t());
  } else {
    // TORCH_CHECK(input.scalar_type() == at::ScalarType::Half ||
    //                 input.scalar_type() == at::ScalarType::BFloat16,
    //             input.scalar_type());
    return runTyped(cutlass::bfloat16_t());
  }
}

TORCH_LIBRARY_IMPL(torchao, CUDA, m) {
  m.impl("torchao::sparse_semi_structured_tile", &_sparse_semi_structured_tile);
}

} // namespace torchao
