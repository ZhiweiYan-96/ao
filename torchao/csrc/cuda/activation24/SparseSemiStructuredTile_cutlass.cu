#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/Dispatch.h>
#include <ATen/Functions.h>
#include <ATen/ScalarOps.h>
#include <ATen/Tensor.h>
#include <ATen/autocast_mode.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/CUDAUtils.h>
#include <c10/cuda/CUDAGuard.h>
#include <torch/library.h>

#include "ComputeSparseTile.cuh"
#include "SparseSemiStructuredPack.cuh"
#include <hip/hip_runtime.h>

namespace torchao {

struct MetadataCutlass {
  // Layout needed to run 2:4 gemms in CUTLASS
  // There is basically a hardware specific value for every
  // 32x32 dense tile (1024 bits). Then these tiles are
  // stored in a Column-Major fashion
  ElementInputE *_meta;
  ElementInputE *_meta_trans;
  int64_t _meta_reordered_sy;
  int64_t _meta_trans_reordered_sx;

  // Define create_compressed_representation
  static std::tuple<at::Tensor, // return value of the function
                    at::Tensor, // packed
                    at::Tensor  // packed_meta
                    >
  create_compressed_representation(int rows, int cols, at::Tensor const &like) {
    TORCH_CHECK(like.scalar_type() == at::ScalarType::Half ||
                like.scalar_type() == at::ScalarType::BFloat16 ||
                like.scalar_type() == at::ScalarType::Float8_e4m3fn);
    auto roundedx = cutlass::round_up(rows, kWarpX);
    auto roundedy = cutlass::round_up(cols, kWarpY);

    // NB: Writing to `packed` tensors in transposed manner
    at::Tensor packed =
        at::empty({roundedx, cutlass::ceil_div(roundedy, 2)}, like.options());
    at::Tensor packed_meta =
        at::empty({roundedx * roundedy / 16},
                  like.options().dtype(at::ScalarType::Byte))
            .view({roundedy / 32, roundedx, 2})
            .permute({1, 2, 0});
    return std::make_tuple(packed, packed, packed_meta);
  }

  // define get_meta_offset
  MetadataCutlass(at::Tensor metaN, at::Tensor metaT, int rows, int cols) {
    _meta = (ElementInputE *)metaN.data_ptr();
    _meta_reordered_sy = metaN.stride(2);
    _meta_trans = (ElementInputE *)metaT.data_ptr();
    _meta_trans_reordered_sx = metaT.stride(2);
  }
  CUTLASS_HOST_DEVICE
  int64_t _get_meta_offset(int warp_row, int thread_row, int warp_col,
                           int thread_col, int64_t stride) const {
    int64_t offset = 0;
    offset += warp_row * 2 + (warp_col / 32) * stride;
    // A single warp is 32x64. The right 32x32 tile is at a different position
    offset += 64 * (thread_row / 32);
    offset += (thread_col / 32) * stride;
    // Top/bottom 16x16 tile
    offset += ((thread_row % 32) / 16) * 4;
    // Left/right 16x16 tile
    offset += ((thread_col % 32) / 16) * 2;
    return offset;
  }

  // Define get_metaN and get_metaT
  CUTLASS_HOST_DEVICE
  ElementInputE *get_metaN(int warp_row, int thread_row, int warp_col,
                           int thread_col) const {
    return _meta + _get_meta_offset(warp_row, thread_row, warp_col, thread_col,
                                    _meta_reordered_sy);
  }
  CUTLASS_HOST_DEVICE
  ElementInputE *get_metaT(int warp_row, int thread_row, int warp_col,
                           int thread_col) const {
    return _meta_trans + _get_meta_offset(warp_col, thread_col, warp_row,
                                          thread_row, _meta_trans_reordered_sx);
  }
};

template <typename KT, typename Metadata, typename Algorithm>
__global__ void __launch_bounds__(32 /* num_threads */, 20)
    sparse_semi_structured_tile_kernel(typename KT::Params p, Metadata metadata,
                                       Algorithm algo) {
  KT::sparse_semi_structured_tile_kernel(p, metadata, algo);
}

template <typename Element, typename MetadataFormat>
std::tuple<at::Tensor, at::Tensor, at::Tensor, at::Tensor, at::Tensor>
sparse_semi_structured_tile_typed(const at::Tensor input,
                                  std::string algorithm) {

  printf("sparse_semi_structured_tile_typed... \n");
  using KT = KernelTypes<Element>;
  std::optional<at::cuda::CUDAGuard> device_guard;
  if (!input.is_meta()) {
    device_guard.emplace(input.device());
  }

  TORCH_CHECK(input.dim() == 2, "Can only sparsify 2d tensors");
  TORCH_CHECK(input.stride(1) == 1, "Can only sparsify contiguous tensors. "
                                    "Sparsify the transpose otherwise.");

  auto rows = input.size(0);
  auto cols = input.size(1);

  auto [compressed, packed, packed_meta_reordered] =
      MetadataFormat::create_compressed_representation(rows, cols, input);
  auto [compressed_trans, packed_trans, packed_trans_meta_reordered] =
      MetadataFormat::create_compressed_representation(cols, rows, input);
  TORCH_CHECK(input.size(1) % 32 == 0,
              "Number of cols should be multiple of 32");

  typename KT::Params p;
  p.input = (Element const *)input.data_ptr();
  p.input_s0 = input.stride(0);
  p.input_dim0 = input.size(0);
  p.input_dim1 = input.size(1);

  p.packed = (Element *)packed.data_ptr();
  p.packed_stride = packed.stride(0);
  p.packed_trans = (Element *)packed_trans.data_ptr();
  p.packed_trans_stride = packed_trans.stride(0);

  MetadataFormat metadata = MetadataFormat(
      packed_meta_reordered, packed_trans_meta_reordered, rows, cols);
  at::Tensor threads_masks = at::empty(
      {p.getBlocksGrid().x * p.getThreadsGrid().x,
       p.getBlocksGrid().y * p.getThreadsGrid().y, sizeof(p.threads_masks[0])},
      input.options().dtype(at::ScalarType::Byte));
  p.threads_masks = (uint64_t *)threads_masks.data_ptr();

  printf("launching kernel ... \n");
  bool kernel_launched = false;
  auto launchKernel = [&](auto algo, std::string const &algo_name) {
    if (algo_name == algorithm) {
      kernel_launched = true;
      if (input.is_meta()) {
        return;
      }
      size_t smem_bytes = 0;
      sparse_semi_structured_tile_kernel<KT>
          <<<p.getBlocksGrid(), p.getThreadsGrid(), smem_bytes,
             at::cuda::getCurrentCUDAStream()>>>(p, metadata, algo);
    }
  };
  named_algorithms(launchKernel);
  TORCH_CHECK(kernel_launched, "Unknown algorithm \"", algorithm, "\"");
  C10_CUDA_KERNEL_LAUNCH_CHECK();
  return std::make_tuple(compressed, packed_meta_reordered, compressed_trans,
                         packed_trans_meta_reordered, threads_masks);
}

// <packed, packed_meta_reordered, packed_trans, packed_trans_meta_reorderd,
// threads_masks>
std::tuple<at::Tensor, at::Tensor, at::Tensor, at::Tensor, at::Tensor>
_sparse_semi_structured_tile(const at::Tensor &input,
                             std::string_view algorithm, bool use_cutlass) {
  std::string algo(algorithm.data(), algorithm.size());

  printf("Start debugging here ...\n");

  auto runTyped = [&](auto type) {
    using ElementT = decltype(type);
    return sparse_semi_structured_tile_typed<ElementT, MetadataCutlass>(input,
                                                                        algo);
  };

  if (input.scalar_type() == at::ScalarType::Half) {
    return runTyped(cutlass::half_t());
  } else if (input.scalar_type() == at::ScalarType::Float8_e4m3fn) {
    return runTyped(cutlass::float_e4m3_t());
  } else {
    // TORCH_CHECK(input.scalar_type() == at::ScalarType::Half ||
    //                 input.scalar_type() == at::ScalarType::BFloat16,
    //             input.scalar_type());
    return runTyped(cutlass::bfloat16_t());
  }
}

TORCH_LIBRARY_IMPL(torchao, CUDA, m) {
  m.impl("torchao::sparse_semi_structured_tile", &_sparse_semi_structured_tile);
}

} // namespace torchao
